
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <iomanip>
#include <vector>
#include <random>
#include <sstream>

using namespace std;

__global__ void add(int *a, int *b, int *c, int n){
	int index = threadIdx.x + blockIdx.x*blockDim.x;

	c[index] = a[index] + b[index];
}

__global__ void print(int *a){
 	printf("%d \n", blockIdx.x);
}

//host ptr, device ptr
template<typename T>
void init_host_device( T* &host_ptr, T* &device_ptr, int size ){
	host_ptr = (T*) malloc( size );
	hipMalloc( (void**)& device_ptr, size );
}

//print vector 
template<typename T>
void print( vector<T>& vec ){
	for (int i = 0; i < vec.size(); ++i){
		cout << vec[i];
	}
	cout << endl;
}

//print vector 
template<typename T>
void print( T*& vec, int size ){
	for (int i = 0; i < size; ++i){
		cout << setw(6) << vec[i];
	}
	cout << endl;
}

//host ptr, device ptr
template<typename T>
void clean_host_device( T* &host_ptr, T* &device_ptr ){
	free( host_ptr );
	hipFree( device_ptr );
}

/*double Random( int High, int Low )
{
    //se usa la funcion time(NULL) para no tener siempre la misma secuencia de aleatorios
    srand( ( unsigned int )time( NULL ) );
    //retorna el numero aleatorio
    return ( (double)rand()/RAND_MAX) * (High - Low) + Low;
}
*/

//print vector 
template<typename T>
void init_rand( T*& vec, int size, int low, int high, int type ){
	//srand
	if( type == 0 ){
		srand( time( NULL ) ); 
		
		for (int i = 0; i < size; ++i){
			vec[i] = rand()%(high-low) + low ;
		}
	}
	//uniform
	else if( type == 1 ){		
		default_random_engine rng(random_device{}()); 		
		uniform_int_distribution<T> dist( low, high );

		for (int i = 0; i < size; ++i){
			vec[i] = dist(rng) ;
		}	
	}
}

template <typename T>
inline void str2num(string str, T& num){
	if ( ! (istringstream(str) >> num) ) num = 0;
}

void test( int N, int M, int type ){
	
	int *a,*b,*c;					// host copies of a,b,c
	int *d_a, *d_b, *d_c;		// device copies of a,b,c
	int size = N * sizeof(int);

	// Allocate space for device copies of a,b,c
	// Alloc space for host copies of a,b,c and setup input
	init_host_device( a, d_a, size);
	init_host_device( b, d_b, size);
	init_host_device( c, d_c, size);

	init_rand( a, N, 0, N, type );
	init_rand( b, N, 0, N*N, type );

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);  // Args: Dir. destino, Dir. origen, tamano de dato, sentido del envio
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<(N+M-1)/M,M>>> (d_a, d_b, d_c, N);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	// print( a, N );
	// print( b, N );
	// print( c, N );

	// Cleanup
	clean_host_device( a, d_a );
	clean_host_device( b, d_b );
	clean_host_device( c, d_c );

}

int main( int argc, char** argv ){

	int N , M ;

	string par = argv[1]; str2num( par, N); 
	par = argv[2]; str2num( par, M);

	time_t timer = time(0);	

	test( N, M, 0 );

	time_t timer2 = time(0);
	cout <<"Tiempo total: " << difftime(timer2, timer) << endl;
	
	timer = time(0);	
	
	test( N, M, 1 );
	
	timer2 = time(0);
	
	cout <<"Tiempo total: " << difftime(timer2, timer) << endl;

	return 0;
}